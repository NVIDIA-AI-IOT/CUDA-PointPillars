#include "hip/hip_runtime.h"
/*
 * SPDX-FileCopyrightText: Copyright (c) 2021 NVIDIA CORPORATION & AFFILIATES. All rights reserved.
 * SPDX-License-Identifier: Apache-2.0
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 * http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include "kernel.h"

__global__ void generateVoxels_random_kernel(float *points, size_t points_size,
        float min_x_range, float max_x_range,
        float min_y_range, float max_y_range,
        float min_z_range, float max_z_range,
        float pillar_x_size, float pillar_y_size, float pillar_z_size,
        int grid_y_size, int grid_x_size,
        unsigned int *mask, float *voxels)
{
  int point_idx = blockIdx.x * blockDim.x + threadIdx.x;
  if(point_idx >= points_size) return;

  float4 point = ((float4*)points)[point_idx];

  if(point.x<min_x_range||point.x>=max_x_range
    || point.y<min_y_range||point.y>=max_y_range
    || point.z<min_z_range||point.z>=max_z_range) return;

  int voxel_idx = floorf((point.x - min_x_range)/pillar_x_size);
  int voxel_idy = floorf((point.y - min_y_range)/pillar_y_size);
  unsigned int voxel_index = voxel_idy * grid_x_size
                            + voxel_idx;

  unsigned int point_id = atomicAdd(&(mask[voxel_index]), 1);

  if(point_id >= POINTS_PER_VOXEL) return;
  float *address = voxels + (voxel_index*POINTS_PER_VOXEL + point_id)*4;
  atomicExch(address+0, point.x);
  atomicExch(address+1, point.y);
  atomicExch(address+2, point.z);
  atomicExch(address+3, point.w);
}

hipError_t generateVoxels_random_launch(float *points, size_t points_size,
        float min_x_range, float max_x_range,
        float min_y_range, float max_y_range,
        float min_z_range, float max_z_range,
        float pillar_x_size, float pillar_y_size, float pillar_z_size,
        int grid_y_size, int grid_x_size,
        unsigned int *mask, float *voxels,
        hipStream_t stream)
{
  int threadNum = THREADS_FOR_VOXEL;
  dim3 blocks((points_size+threadNum-1)/threadNum);
  dim3 threads(threadNum);
  generateVoxels_random_kernel<<<blocks, threads, 0, stream>>>
    (points, points_size,
        min_x_range, max_x_range,
        min_y_range, max_y_range,
        min_z_range, max_z_range,
        pillar_x_size, pillar_y_size, pillar_z_size,
        grid_y_size, grid_x_size,
        mask, voxels);
  hipError_t err = hipGetLastError();
  return err;
}

__global__ void generateVoxelsList_kernel(float *points, size_t points_size,
        float min_x_range, float max_x_range,
        float min_y_range, float max_y_range,
        float min_z_range, float max_z_range,
        float pillar_x_size, float pillar_y_size, float pillar_z_size,
        int grid_y_size, int grid_x_size,
        unsigned int *mask, int *voxelsList)
{
  int point_idx = blockIdx.x * blockDim.x + threadIdx.x;
  if(point_idx >= points_size) return;

  float4 point = ((float4*)points)[point_idx];

  if(point.x<min_x_range||point.x>=max_x_range
    || point.y<min_y_range||point.y>=max_y_range
    || point.z<min_z_range||point.z>=max_z_range)
  {
    voxelsList[point_idx] = -1;
    return;
  }

  int voxel_idx = floorf((point.x - min_x_range)/pillar_x_size);
  int voxel_idy = floorf((point.y - min_y_range)/pillar_y_size);
  unsigned int voxel_index = voxel_idy * grid_x_size
                            + voxel_idx;

  atomicAdd(&(mask[voxel_index]), 1);
  voxelsList[point_idx] = voxel_index;

}

__global__ void generateVoxels_kernel(float *points, size_t points_size,
        int *voxelsList,
        unsigned int *mask, float *voxels)
{
  int point_idx = blockIdx.x * blockDim.x + threadIdx.x;
  if(point_idx >= points_size) return;

  int voxel_index = voxelsList[point_idx];

  if (voxel_index == -1) return;
  int point_id = atomicAdd(&(mask[voxel_index]), 1);

  if(point_id >= POINTS_PER_VOXEL) return;
  float *address = voxels + (voxel_index*POINTS_PER_VOXEL + point_id)*4;
  float4 point = ((float4*)points)[point_idx];
  atomicExch(address+0, point.x);
  atomicExch(address+1, point.y);
  atomicExch(address+2, point.z);
  atomicExch(address+3, point.w);
}

__global__ void generateBaseFeatures_kernel(unsigned int *mask, float *voxels,
        int grid_y_size, int grid_x_size,
        unsigned int *pillar_num,
        float *voxel_features,
        unsigned int *voxel_num,
        unsigned int *voxel_idxs)
{
  unsigned int voxel_idx = blockIdx.x * blockDim.x + threadIdx.x;
  unsigned int voxel_idy = blockIdx.y * blockDim.y + threadIdx.y;

  if(voxel_idx >= grid_x_size ||voxel_idy >= grid_y_size) return;

  unsigned int voxel_index = voxel_idy * grid_x_size
                           + voxel_idx;
  unsigned int count = mask[voxel_index];
  if( !(count>0) ) return;
  count = count<POINTS_PER_VOXEL?count:POINTS_PER_VOXEL;

  unsigned int current_pillarId = 0;
  current_pillarId = atomicAdd(pillar_num, 1);

  if( current_pillarId >= MAX_VOXELS ) return;
  
  voxel_num[current_pillarId] = count;

  uint4 idx = {0, 0, voxel_idy, voxel_idx};
  ((uint4*)voxel_idxs)[current_pillarId] = idx;

  for (int i=0; i<count; i++){
    int inIndex = voxel_index*POINTS_PER_VOXEL + i;
    int outIndex = current_pillarId*POINTS_PER_VOXEL + i;
    ((float4*)voxel_features)[outIndex] = ((float4*)voxels)[inIndex];
  }

  // clear buffer for next infer
  atomicExch(mask + voxel_index, 0);
}

// create 4 channels
hipError_t generateBaseFeatures_launch(unsigned int *mask, float *voxels,
        int grid_y_size, int grid_x_size,
        unsigned int *pillar_num,
        float *voxel_features,
        unsigned int *voxel_num,
        unsigned int *voxel_idxs,
        hipStream_t stream)
{
  dim3 threads = {32,32};
  dim3 blocks = {(grid_x_size + threads.x -1)/threads.x,
                 (grid_y_size + threads.y -1)/threads.y};

  generateBaseFeatures_kernel<<<blocks, threads, 0, stream>>>
      (mask, voxels, grid_y_size, grid_x_size,
       pillar_num,
       voxel_features,
       voxel_num,
       voxel_idxs);
  hipError_t err = hipGetLastError();
  return err;
}

// 4 channels -> 10 channels
__global__ void generateFeatures_kernel(float* voxel_features,
    unsigned int* voxel_num, unsigned int* voxel_idxs, unsigned int *params,
    float voxel_x, float voxel_y, float voxel_z,
    float range_min_x, float range_min_y, float range_min_z,
    float* features)
{
    int pillar_idx = blockIdx.x * WARPS_PER_BLOCK + threadIdx.x/WARP_SIZE;
    int point_idx = threadIdx.x % WARP_SIZE;

    int pillar_idx_inBlock = threadIdx.x/32;
    unsigned int num_pillars = params[0];

    if (pillar_idx >= num_pillars) return;

    __shared__ float4 pillarSM[WARPS_PER_BLOCK][WARP_SIZE];
    __shared__ float4 pillarSumSM[WARPS_PER_BLOCK];
    __shared__ uint4 idxsSM[WARPS_PER_BLOCK];
    __shared__ int pointsNumSM[WARPS_PER_BLOCK];
    __shared__ float pillarOutSM[WARPS_PER_BLOCK][WARP_SIZE][FEATURES_SIZE];

    if (threadIdx.x < WARPS_PER_BLOCK) {
      pointsNumSM[threadIdx.x] = voxel_num[blockIdx.x * WARPS_PER_BLOCK + threadIdx.x];
      idxsSM[threadIdx.x] = ((uint4*)voxel_idxs)[blockIdx.x * WARPS_PER_BLOCK + threadIdx.x];
      pillarSumSM[threadIdx.x] = {0,0,0,0};
    }

    pillarSM[pillar_idx_inBlock][point_idx] = ((float4*)voxel_features)[pillar_idx*WARP_SIZE + point_idx];
    __syncthreads();

    //calculate sm in a pillar
    if (point_idx < pointsNumSM[pillar_idx_inBlock]) {
      atomicAdd(&(pillarSumSM[pillar_idx_inBlock].x),  pillarSM[pillar_idx_inBlock][point_idx].x);
      atomicAdd(&(pillarSumSM[pillar_idx_inBlock].y),  pillarSM[pillar_idx_inBlock][point_idx].y);
      atomicAdd(&(pillarSumSM[pillar_idx_inBlock].z),  pillarSM[pillar_idx_inBlock][point_idx].z);
    }
    __syncthreads();

    //feature-mean
    float4 mean;
    float validPoints = pointsNumSM[pillar_idx_inBlock];
    mean.x = pillarSumSM[pillar_idx_inBlock].x / validPoints;
    mean.y = pillarSumSM[pillar_idx_inBlock].y / validPoints;
    mean.z = pillarSumSM[pillar_idx_inBlock].z / validPoints;

    mean.x  = pillarSM[pillar_idx_inBlock][point_idx].x - mean.x;
    mean.y  = pillarSM[pillar_idx_inBlock][point_idx].y - mean.y;
    mean.z  = pillarSM[pillar_idx_inBlock][point_idx].z - mean.z;


    //calculate offset
    float x_offset = voxel_x / 2 + idxsSM[pillar_idx_inBlock].w * voxel_x + range_min_x;
    float y_offset = voxel_y / 2 + idxsSM[pillar_idx_inBlock].z * voxel_y + range_min_y;
    float z_offset = voxel_z / 2 + idxsSM[pillar_idx_inBlock].y * voxel_z + range_min_z;

    //feature-offset
    float4 center;
    center.x  = pillarSM[pillar_idx_inBlock][point_idx].x - x_offset;
    center.y  = pillarSM[pillar_idx_inBlock][point_idx].y - y_offset;
    center.z  = pillarSM[pillar_idx_inBlock][point_idx].z - z_offset;

    //store output
    if (point_idx < pointsNumSM[pillar_idx_inBlock]) {
      pillarOutSM[pillar_idx_inBlock][point_idx][0] = pillarSM[pillar_idx_inBlock][point_idx].x;
      pillarOutSM[pillar_idx_inBlock][point_idx][1] = pillarSM[pillar_idx_inBlock][point_idx].y;
      pillarOutSM[pillar_idx_inBlock][point_idx][2] = pillarSM[pillar_idx_inBlock][point_idx].z;
      pillarOutSM[pillar_idx_inBlock][point_idx][3] = pillarSM[pillar_idx_inBlock][point_idx].w;

      pillarOutSM[pillar_idx_inBlock][point_idx][4] = mean.x;
      pillarOutSM[pillar_idx_inBlock][point_idx][5] = mean.y;
      pillarOutSM[pillar_idx_inBlock][point_idx][6] = mean.z;

      pillarOutSM[pillar_idx_inBlock][point_idx][7] = center.x;
      pillarOutSM[pillar_idx_inBlock][point_idx][8] = center.y;
      pillarOutSM[pillar_idx_inBlock][point_idx][9] = center.z;

    } else {
      pillarOutSM[pillar_idx_inBlock][point_idx][0] = 0;
      pillarOutSM[pillar_idx_inBlock][point_idx][1] = 0;
      pillarOutSM[pillar_idx_inBlock][point_idx][2] = 0;
      pillarOutSM[pillar_idx_inBlock][point_idx][3] = 0;

      pillarOutSM[pillar_idx_inBlock][point_idx][4] = 0;
      pillarOutSM[pillar_idx_inBlock][point_idx][5] = 0;
      pillarOutSM[pillar_idx_inBlock][point_idx][6] = 0;

      pillarOutSM[pillar_idx_inBlock][point_idx][7] = 0;
      pillarOutSM[pillar_idx_inBlock][point_idx][8] = 0;
      pillarOutSM[pillar_idx_inBlock][point_idx][9] = 0;
    }

    __syncthreads();

    for(int i = 0; i < FEATURES_SIZE; i ++) {
      int outputSMId = pillar_idx_inBlock*WARP_SIZE*FEATURES_SIZE + i* WARP_SIZE + point_idx;
      int outputId = pillar_idx*WARP_SIZE*FEATURES_SIZE + i* WARP_SIZE + point_idx;
      features[outputId] = ((float*)pillarOutSM)[outputSMId] ;
    }

}

hipError_t generateFeatures_launch(float* voxel_features,
    unsigned int * voxel_num,
    unsigned int* voxel_idxs,
    unsigned int *params,
    float voxel_x, float voxel_y, float voxel_z,
    float range_min_x, float range_min_y, float range_min_z,
    float* features,
    hipStream_t stream)
{
    dim3 blocks( (MAX_VOXELS+WARPS_PER_BLOCK-1)/WARPS_PER_BLOCK);
    dim3 threads(WARPS_PER_BLOCK*WARP_SIZE);

    generateFeatures_kernel<<<blocks, threads, 0, stream>>>
      (voxel_features,
      voxel_num,
      voxel_idxs,
      params,
      voxel_x, voxel_y, voxel_z,
      range_min_x, range_min_y, range_min_z,
      features);

    hipError_t err = hipGetLastError();
    return err;
}
